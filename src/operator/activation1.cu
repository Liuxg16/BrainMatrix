#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2015 by Contributors
 * \file activation1.cu
 * \brief
 * \author Xianggen Liu
*/
#include "./activation1-inl.h"
#include "./mshadow_op.h"
#if MXNET_USE_CUDNN == 1
#include "./cudnn_activation1-inl.h"
#endif

namespace mxnet {
namespace op {
template<>
Operator *CreateOp<gpu>(Activation1Param param) {
  // SoftReLU not supported by CUDNN yet
  if (param.act_type == activation1::kSoftReLU)
      return new Activation1Op<gpu, mshadow_op::softrelu, mshadow_op::softrelu_grad>();

#if MXNET_USE_CUDNN == 1
  return new CuDNNActivation1Op(param);
#else
  switch(param.act_type) {
    case activation1::kReLU:
      return new Activation1Op<gpu, mshadow_op::relu, mshadow_op::relu_grad>();
    case activation1::kSigmoid:
      return new Activation1Op<gpu, mshadow_op::sigmoid, mshadow_op::sigmoid_grad>();
    case activation1::kTanh:
      return new Activation1Op<gpu, mshadow_op::tanh, mshadow_op::tanh_grad>();
    default:
      LOG(FATAL) << "unknown activation1";
      return NULL;
  }
#endif  // MXNET_USE_CUDNN
}
}  // op
}  // namespace mxnet

